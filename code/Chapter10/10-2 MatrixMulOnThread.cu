#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include "../common.h"

__global__ void MatrixMulKernel01(float* A, float* B, float* C, int width)
{
    	int tx = threadIdx.x;
    	int bx = blockDim.x;
	int idx=bx*width+tx;
	int row=idx/width;
	int col=idx%width;
	if(row<width && col<width){
		float Pvalue = 0;
		for (int k = 0; k<width; k++){
			float Mdelement = A[row*width + k];
			float Ndelement = B[k*width + col];
			Pvalue += Mdelement * Ndelement;
         }
		C[row*width + col] = Pvalue;
	  }
}
__global__ void MatrixMulKernel02(float* A, float* B, float* C, int width)
{
	int tx = threadIdx.x;
	int bx=blockIdx.x;
    for (bx = blockIdx.x; bx < width; bx += gridDim.x){
		for (tx  =  threadIdx.x; tx < width; tx += blockDim.x){
			float Pvalue = 0;
			for (int k = 0; k < width; k++){
				float Mdelement = A[bx * width + k];
				float Ndelement = B[k * width + tx];
				Pvalue += Mdelement * Ndelement;
            }
			C[bx * width + tx] = Pvalue;	
		}
	}
}

void MatrixMulOnHost(float *A, float *B, float *C, int width)
{
    int i, j, k;
    double temp = 0.0;
    float *B1;
    B1 = (float *)malloc(sizeof(float) * width * width);
    for (int i = 0; i < width; i++)
    {
         for(int j = 0; j < width; j++)
         {
              B1[i * width + j] = B[j * width +i];
          }
    }
    for (i = 0; i < width; i++)
    {
         for (j = 0; j < width; j++)
         {
             temp = 0.0;
             for (k = 0; k < width; k++)
             {
                  temp += A[i * width + k] * B1[j * width +k];
             }    
             C[i * width + j] = temp;
           }
    }
    free (B1);
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match)
        printf("Results match.\n\n");
    else
        printf("Results do not match.\n\n");
}

int main(void){
int dev = 0;
hipDeviceProp_t deviceProp;
CHECK(hipGetDeviceProperties(&deviceProp, dev));
printf("using Device %d: %s\n", dev, deviceProp.name);
CHECK(hipSetDevice(dev));
    //矩阵元素
int Width = 1 << 10 ;
int size = Width*Width*sizeof(float); 

float *M, *N, *P, *gpuRef01, *gpuRef02; 
M = (float *)malloc(size);
N = (float *)malloc(size); 
P = (float *)malloc(size);
gpuRef01 = (float *)malloc(size);
gpuRef02 = (float *)malloc(size);
double iStart = seconds();
//初始化示例数据
for (int i = 0; i<Width; i++){
for (int j = 0; j<Width; j++){
      M[i*Width+j] = 2.0 ;
      N[i*Width+j] = 3.0 ;			
}
}
double iElaps = seconds() - iStart;
printf("initialization: \t %f sec\n", iElaps);
//矩阵乘在主机端运行
iStart = seconds();
MatrixMulOnHost_3(M, N, P, Width);
iElaps = seconds() - iStart;
printf("MatrixMulOnHost_3 : \t %f sec\n", iElaps);

float *Md, *Nd, *Pd;
hipMalloc((void**)&Md, size);
hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
hipMalloc((void**)&Nd, size);
hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
hipMalloc((void**)&Pd, size);

dim3 block(1024);
dim3 grid((Width + block.x - 1) / block.x);                  
//矩阵乘在设备端运行
iStart = seconds();
MatrixMulKernel01<<<grid,block>>>(Md, Nd, Pd, Width);
iElaps = seconds() - iStart;
printf("MatrixMulKernel01 on device  <<< %d,%d>>>:\t %f sec\n",grid.x, block.x, iElaps);
hipMemcpy(gpuRef01, Pd, size, hipMemcpyDeviceToHost);
checkResult(P, gpuRef01, Width);

iStart = seconds();
MatrixMulKernel02<<<grid , block>>>(Md, Nd, Pd, Width);
iElaps = seconds() - iStart;
printf("MatrixMulKernel02 on device  <<< %d,%d>>>:\t %f sec\n",grid.x, block.x, iElaps);
hipMemcpy(gpuRef02, Pd, size, hipMemcpyDeviceToHost);
checkResult(P, gpuRef02, Width);

hipFree(M);
hipFree(N);
hipFree(P);
hipFree(gpuRef1);
hipFree(gpuRef2);
hipFree(Md);
hipFree(Nd);
hipFree(Pd);
return 0;
}
