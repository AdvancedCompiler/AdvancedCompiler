#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include "../common.h"

__global__ void MatrixMulKernel01(float* Md, float* Nd, float* Pd, int width)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int bx = threadIdx.y + blockIdx.y * blockDim.y;
	int idx=bx*width+tx;
	int row=idx/width;
	int col=idx%width;
	
	if(row<width && col<width){
		float Pvalue = 0;
		for (int k = 0; k<width; k++)
		{
			float Mdelement = Md[row*width + k];
			float Ndelement = Nd[k*width + col];
			Pvalue += Mdelement * Ndelement;
                                 }
		Pd[row*width + col] = Pvalue;
	  }
}

void MatrixMulOnHost_3(float *A, float *B, float *C, int width)
{
    int i, j, k;
    double temp = 0.0;
    float *B1;
    B1 = (float *)malloc(sizeof(float) * width * width);
    for (int i = 0; i < width; i++)
    {
         for(int j = 0; j < width; j++)
         {
              B1[i * width + j] = B[j * width +i];
          }
    }
    for (i = 0; i < width; i++)
    {
         for (j = 0; j < width; j++)
         {
             temp = 0.0;
             for (k = 0; k < width; k++)
             {
                  temp += A[i * width + k] * B1[j * width +k];
             }    
             C[i * width + j] = temp;
           }
    }
    free (B1);
}

//结果检查
void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match)
        printf("Results match.\n\n");
    else
        printf("Results do not match.\n\n");
}


int main(void){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
	
    int Width = 1 << 10 ;
    int size = Width*Width*sizeof(float); 

    float *M, *N, *P, *gpuRef; 
    M = (float *)malloc(size);
    N = (float *)malloc(size); 
    P = (float *)malloc(size);
    gpuRef = (float *)malloc(size);
    
    double iStart = seconds();
    for (int i = 0; i<Width; i++){
        for (int j = 0; j<Width; j++){
            M[i*Width+j] = 2.0 ;
            N[i*Width+j] = 3.0 ;			
        }
    }
    double iElaps = seconds() - iStart;
    printf("initialization: \t %f sec\n", iElaps);

    iStart = seconds();
    MatrixMulOnHost_3(M, N, P, Width);
    iElaps = seconds() - iStart;
    printf("MatrixMulOnHost_3 : \t %f sec\n", iElaps);

    float *Md, *Nd, *Pd;
    hipMalloc((void**)&Md, size);
    hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
    hipMalloc((void**)&Nd, size);
    hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
    hipMalloc((void**)&Pd, size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx,dimy);
    dim3 grid((Width + block.x - 1) / block.x,  (Width + block.y - 1) / block.y);

    iStart = seconds();
    MatrixMulKernel03<<<grid,block>>>(Md, Nd, Pd, Width);
    iElaps = seconds() - iStart;
    printf("MatrixMulKernel03 on device  <<< (%d,%d), (%d,%d)>>>:\t %f sec\n",grid.x, grid.y, block.x, block.y, iElaps);
    hipMemcpy(gpuRef, Pd, size, hipMemcpyDeviceToHost);

    checkResult(P, gpuRef, Width);                                        

    hipFree(M);
    hipFree(N);
    hipFree(P);
    hipFree(gpuRef);
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);

    return 0;
}
