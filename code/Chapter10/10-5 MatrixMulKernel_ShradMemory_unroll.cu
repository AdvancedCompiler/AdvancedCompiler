#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include "../common.h"


//共享内存优化核函数
__global__ void MatrixMulKernel_ShradMemory(float* Md, float* Nd, float* Pd, int width){
	extern __shared__ float data[];
	const int tid = threadIdx.x;
	const int row = blockIdx.x;
	int i, j;

	for(i = tid; i < width; i += blockDim.x){
		data[i] = Md[row * width + i];
	}
	__syncthreads();
	double tmp = 0.0;
	for(j = tid; j < width; j += blockDim.x){
		tmp = 0.0;
		for(i = 0; i < width; i++){
			tmp += data[i] * Nd[i * width + j];
		}
		Pd[row * width + j] = tmp;
	}
}


//循环展开核函数
__global__ void MatrixMulKernel_ShradMemory_unroll2(float* Md, float* Nd, float* Pd, int width){
	extern __shared__ float data[];
	const int tid = threadIdx.x;
	const int row = blockIdx.x;
	int i, j;

	for(i = tid; i < width; i += blockDim.x){
		data[i] = Md[row * width + i];
	}
	__syncthreads();
	double tmp = 0.0;
	for(j = tid; j < width; j += blockDim.x){
		tmp = 0.0;
		for(i = 0; i < width/2; i++){
			tmp += data[i] * Nd[i * width + j];
			tmp += data[i+1] * Nd[(i+1) * width + j]; 
			
		}
		Pd[row * width + j] = tmp;
	}
}

//主机端矩阵乘法运算
void MatrixMulOnHost_3(float *A, float *B, float *C, int width)
{
    int i, j, k;
    double temp = 0.0;
    float *B1;
    B1 = (float *)malloc(sizeof(float) * width * width);
    for (int i = 0; i < width; i++){
         for(int j = 0; j < width; j++){
              B1[i * width + j] = B[j * width +i];
          }
    }
    for (i = 0; i < width; i++){
         for (j = 0; j < width; j++){
             temp = 0.0;
             for (k = 0; k < width; k++){
                  temp += A[i * width + k] * B1[j * width +k];
             }    
             C[i * width + j] = temp;
           }
    }
    free (B1);
}

//主机端矩阵乘法运算展开循环
void MatrixMulOnHost_3_unroll2(float *A, float *B, float *C, int width)
{
    int i, j, k;
    double temp = 0.0;
    float *B1;
    B1 = (float *)malloc(sizeof(float) * width * width);
    for (int i = 0; i < width; i++){
         for(int j = 0; j < width; j++){
              B1[i * width + j] = B[j * width +i];
          }
    }
    for (i = 0; i < width; i++){
         for (j = 0; j < width; j++){
             temp = 0.0;
             for (k = 0; k < width/2; k++){
                  temp += A[i * width + k] * B1[j * width +k];
                  temp += A[i * width + k+1] * B1[j * width +k+1];
             }    
             C[i * width + j] = temp;
           }
    }
    free (B1);
}

//结果检查
void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i = 0; i < N; i++){
        if (abs(hostRef[i] - gpuRef[i]) > epsilon){
            match = 0;
            printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
            break;
        }
    }
    if (match)
        printf("Results match.\n\n");
    else
        printf("Results do not match.\n\n");
}

int main(void)
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
	                          
	int Width = 1<<11;
	int size = Width*Width*sizeof(float); 
               
    float *M, *N, *P, *gpuRef1, *gpuRef2; 
    M = (float *)malloc(size);
    N = (float *)malloc(size); 
    P = (float *)malloc(size);
    gpuRef1 = (float *)malloc(size);
	gpuRef2 = (float *)malloc(size);
                            
    double iStart = seconds();
    //初始化示例数据
	for (int i = 0; i<Width; i++){
        for (int j = 0; j<Width; j++){
            M[i*Width+j] = 3.0 ;
            N[i*Width+j] = 3.0 ;			
		}
	}
    double iElaps = seconds() - iStart;
    printf("initialization: \t %f sec\n", iElaps);
                             
/*------------------主机端-------------------------------------------------------*/

/*----------------------MatrixMulOnHost_3--------------------------------*/    
    iStart = seconds();
    MatrixMulOnHost_3(M, N, P, Width);
    iElaps = seconds() - iStart;
    printf("MatrixMulOnHost_3 : \t %f sec\n", iElaps);

    iStart = seconds();
    MatrixMulOnHost_3_unroll2(M, N, P, Width);
    iElaps = seconds() - iStart;
    printf("MatrixMulOnHost_3_unroll2 : \t %f sec\n", iElaps);

/*------------------设备端-------------------------------------------------------- */	
	float *Md, *Nd, *Pd;
	hipMalloc((void**)&Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&Pd, size);
                
	//int dimx = 32;
    //int dimy = 32;
    //dim3 block(dimx,dimy);
    //dim3 grid((Width + block.x - 1) / block.x,  (Width + block.y - 1) / block.y);                  
	dim3 block(1024);
    dim3 grid((Width + block.x - 1) / block.x);                 

/*----------------------MatrixMulKernel_ShradMemory----------------------*/
	iStart = seconds();
	MatrixMulKernel_ShradMemory<<<grid,block,sizeof(float)*(Width)>>>(Md, Nd, Pd, Width);
	iElaps = seconds() - iStart;
	printf("MatrixMulKernel_ShradMemory on device  <<< %d, %d>>>:\t %f sec\n",grid.x, block.x, iElaps);
 	hipMemcpy(gpuRef1, Pd, size, hipMemcpyDeviceToHost);
    checkResult(P, gpuRef1, Width);

/*------------------MatrixMulKernel_ShradMemory_unroll2-------------------*/
	iStart = seconds();
	MatrixMulKernel_ShradMemory_unroll2k<<<grid,block,sizeof(float)*(Width)>>>(Md, Nd, Pd, Width);
	iElaps = seconds() - iStart;
	printf("MatrixMulKernel_ShradMemory_unroll2 on device  <<< %d, %d>>>:\t %f sec\n",grid.x, block.x, iElaps);
 	hipMemcpy(gpuRef2, Pd, size, hipMemcpyDeviceToHost);
    checkResult(P, gpuRef2, Width);
                            
    hipFree(M);
    hipFree(N);
    hipFree(P);
    hipFree(gpuRef1);
	hipFree(gpuRef2);
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);
	hipDeviceReset();
	
	return 0;
}