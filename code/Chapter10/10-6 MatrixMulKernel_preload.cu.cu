#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include "../common.h"

//block线程循环优化
__global__ void MatrixMulKernel02(float* A, float* B, float* C, int width)
{
	int tx = threadIdx.x;
	int bx=blockIdx.x;
    for (bx = blockIdx.x; bx < width; bx += gridDim.x){
		for (tx  =  threadIdx.x; tx < width; tx += blockDim.x){
			float Pvalue = 0;
			for (int k = 0; k < width; k++){
				float Mdelement = A[bx * width + k];
				float Ndelement = B[k * width + tx];
				Pvalue += Mdelement * Ndelement;
            }
			C[bx * width + tx] = Pvalue;	
		}
	}
}

//数据预取核函数
__global__ void MatrixMulKernel02_preload(float* A, float* B, float* C, int width)
{
	int tx = threadIdx.x;
	int bx=blockIdx.x;
	extern __device__ float data1[256];
	extern __device__ float data2[256];
	for(int i = tx; i < width; i += blockDim.x){
		data1[i] = A[bx * width + i];
		data2[i] = B[i * width + tx];
	}
   	 for (bx = blockIdx.x; bx < width; bx += gridDim.x){
		for (tx =  threadIdx.x; tx < width; tx += blockDim.x){
			float Pvalue1 = 0;
			for (int k = 0; k < width; k++){
				Pvalue1 += data1[k] * data2[k];
             }
				C[bx * width + tx] = Pvalue1;			
		}
	}
}

//主机端矩阵乘法运算
void MatrixMulOnHost_3(float *A, float *B, float *C, int width)
{
    int i, j, k;
    double temp = 0.0;
    float *B1;
    B1 = (float *)malloc(sizeof(float) * width * width);
    for (int i = 0; i < width; i++){
         for(int j = 0; j < width; j++){
              B1[i * width + j] = B[j * width +i];
          }
    }
    for (i = 0; i < width; i++){
         for (j = 0; j < width; j++){
             temp = 0.0;
             for (k = 0; k < width; k++){
                  temp += A[i * width + k] * B1[j * width +k];
             }    
             C[i * width + j] = temp;
           }
    }
    free (B1);
}

//结果检查
void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i = 0; i < N; i++){
        if (abs(hostRef[i] - gpuRef[i]) > epsilon){
            match = 0;
            printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
            break;
        }
    }
    if (match)
        printf("Results match.\n\n");
    else
        printf("Results do not match.\n\n");
}

int main(void)
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
	                          
	int Width = 1<<11;
	int size = Width*Width*sizeof(float); 
               
    float *M, *N, *P, *gpuRef1, *gpuRef2; 
    M = (float *)malloc(size);
    N = (float *)malloc(size); 
    P = (float *)malloc(size);
    gpuRef1 = (float *)malloc(size);
	gpuRef2 = (float *)malloc(size);
                            
    double iStart = seconds();
    //初始化示例数据
	for (int i = 0; i<Width; i++){
        for (int j = 0; j<Width; j++){
            M[i*Width+j] = 3.0 ;
            N[i*Width+j] = 3.0 ;			
		}
	}
    double iElaps = seconds() - iStart;
    printf("initialization: \t %f sec\n", iElaps);
                             
/*------------------主机端-------------------------------------------------------*/

/*----------------------MatrixMulOnHost_3--------------------------------*/    
    iStart = seconds();
    MatrixMulOnHost_3(M, N, P, Width);
    iElaps = seconds() - iStart;
    printf("MatrixMulOnHost_3 : \t %f sec\n", iElaps);

/*------------------设备端-------------------------------------------------------- */	
	float *Md, *Nd, *Pd;
	hipMalloc((void**)&Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&Pd, size);
                              
	dim3 block(1024);
    dim3 grid((Width + block.x - 1) / block.x);                 

/*----------------------MatrixMulKernel02----------------------*/
	iStart = seconds();
	MatrixMulKernel02<<<grid,block,sizeof(float)*(Width)>>>(Md, Nd, Pd, Width);
	iElaps = seconds() - iStart;
	printf("MatrixMulKernel02 on device  <<< %d, %d>>>:\t %f sec\n",grid.x, block.x, iElaps);
 	hipMemcpy(gpuRef1, Pd, size, hipMemcpyDeviceToHost);
    checkResult(P, gpuRef1, Width);

/*------------------MatrixMulKernel02_preload-------------------*/
	iStart = seconds();
	MatrixMulKernel02_preload<<<grid,block,sizeof(float)*(Width)>>>(Md, Nd, Pd, Width);
	iElaps = seconds() - iStart;
	printf("MatrixMulKernel02_preload on device  <<< %d, %d>>>:\t %f sec\n",grid.x, block.x, iElaps);
 	hipMemcpy(gpuRef2, Pd, size, hipMemcpyDeviceToHost);
    checkResult(P, gpuRef2, Width);
                            
    hipFree(M);
    hipFree(N);
    hipFree(P);
    hipFree(gpuRef1);
	hipFree(gpuRef2);
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);
	hipDeviceReset();
	
	return 0;
}